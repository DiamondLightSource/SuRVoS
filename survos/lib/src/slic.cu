#include "hip/hip_runtime.h"

#include "slic.cuh"

#define DLIMIT FLT_MAX
#define i(a, b, c) ((c) * shape.y * shape.x + (b) * shape.x + (a))
#define max_cu(a,b)  ({ __typeof__ (a) _a = (a); __typeof__ (b) _b = (b); _a > _b ? _a : _b; })

__global__
void initSupervoxels(const float *data,
                     SLICClusterCenter* centers,
                     const int tClusters,
                     const int3 nClusters,
                     const int3 sp_shape,
                     const int3 window,
                     const int3 shape)
{
    size_t lidx  = threadIdx.x + (blockIdx.x * blockDim.x);

    if ( lidx >= tClusters )
        return;

    int3 idx;
    int plane = nClusters.y * nClusters.x;
    idx.z = lidx / plane;
    int aux = lidx % plane;
    idx.y = aux / nClusters.x;
    idx.x = aux % nClusters.x;

    int x = idx.x * sp_shape.x + sp_shape.x / 2;
    int y = idx.y * sp_shape.y + sp_shape.y / 2;
    int z = idx.z * sp_shape.z + sp_shape.z / 2;

    int u, v, w, cx = x, cy = y, cz = z, cux, cuy, cuz;
    float minGradient = DLIMIT, gradient, dx, dy, dz;

    for ( u = -window.x; u <= window.x; u++ )
    {
        for ( v = -window.y; v <= window.y; v++ )
        {
            for ( w = -window.z; w <= window.z; w++ )
            {
                cux = x+u; cuy = y+v; cuz = z+w;
                if ( cux < 1 || cux > shape.x-2 ||
                     cuy < 1 || cuy > shape.y-2 ||
                     cuz < 1 || cuz > shape.z-2 )
                {
                    continue;
                }

                dx = data[i(cux+1, cuy, cuz)] - data[i(cux-1, cuy, cuz)];
                dy = data[i(cux, cuy+1, cuz)] - data[i(cux, cuy-1, cuz)];
                dz = data[i(cux, cuy, cuz+1)] - data[i(cux, cuy, cuz-1)];

                gradient = dx*dx + dy*dy + dz*dz;

                if ( gradient < minGradient )
                {
                    minGradient = gradient;
                    cx = cux;
                    cy = cuy;
                    cz = cuz;
                }
            }
        }
    }

    centers[lidx].f = data[i(cx, cy, cz)];
    centers[lidx].x = cx;
    centers[lidx].y = cy;
    centers[lidx].z = cz;
}

__global__
void assignSupervoxels(const float *data,
                       const SLICClusterCenter* centers,
                       int *labels,
                       const float compactness,
                       const int tClusters,
                       const float3 spacing,
                       const int3 nClusters,
                       const int3 sp_shape,
                       const int3 im_shape)
{
    int3 idx;

    size_t gidx  = threadIdx.x + (blockIdx.x * blockDim.x);

    if ( gidx >= im_shape.x * im_shape.y * im_shape.z )
        return;

    int plane = (im_shape.y * im_shape.x);

    idx.z = gidx / plane;
    int aux = gidx % plane;
    idx.y = aux / im_shape.x;
    idx.x = aux % im_shape.x;

    int cz = idx.z / sp_shape.z;
    if ( cz >= nClusters.z ) { cz = nClusters.z - 1; }
    int cy = idx.y / sp_shape.y;
    if ( cy >= nClusters.y ) { cy = nClusters.y - 1; }
    int cx = idx.x / sp_shape.x;
    if ( cx >= nClusters.x ) { cx = nClusters.x - 1; }

    float minDist = DLIMIT;
    int minIdx = cz * cy * cx;

    int R = 2;

    for ( int k = -R; k < 1+R; k++ ) {
        int ccz = cz + k;
        if ( ccz < 0 || ccz >= nClusters.z ) {
            continue;
        }
        for ( int i = -R; i < 1+R; i++ ) {
            int ccy = cy + i;
            if ( ccy < 0 || ccy >= nClusters.y ) {
                continue;
            }
            for ( int j = -R; j < 1+R; j++ ) {
                int ccx = cx + j;
                if ( ccx < 0 || ccx >= nClusters.x ) {
                    continue;
                }
                int cidx = ccz * nClusters.y * nClusters.x + ccy * nClusters.x + ccx;
                if ( cidx >= tClusters || centers[cidx].z < 0 ) {
                    continue;
                }
                float dist_g = data[gidx] - centers[cidx].f;
                float dx = (idx.x - centers[cidx].x) * spacing.x;
                float dy = (idx.y - centers[cidx].y) * spacing.y;
                float dz = (idx.z - centers[cidx].z) * spacing.z;
                float dist_s = dx*dx + dy*dy + dz*dz;
                float dist = dist_g * dist_g + compactness * dist_s;

                if ( dist < minDist ) {
                    minDist = dist;
                    minIdx = cidx;
                }
            }
        }
    }
    labels[gidx] = minIdx;
}

__global__
void updateSupervoxels(const float *data,
                       const int *labels,
                       SLICClusterCenter* centers,
                       const int tClusters,
                       const int3 nClusters,
                       const int3 spshape,
                       const int3 imshape)
{
    size_t lidx  = threadIdx.x + (blockIdx.x * blockDim.x);

    if ( lidx >= tClusters )
        return;

    float cz = centers[lidx].z, cy = centers[lidx].y, cx = centers[lidx].x;

    if ( cz == -1 )
        return;

    int kinit, kend, jinit, jend, iinit, iend;

    float ratio = 2.0f;
    kinit = cz - spshape.z * ratio;
    if ( kinit < 0 ) kinit = 0;
    kend = cz + spshape.z * ratio;
    if ( kend >= imshape.z ) kend = imshape.z - 1;

    iinit = cy - spshape.y * ratio;
    if ( iinit < 0 ) iinit = 0;
    iend = cy + spshape.y * ratio;
    if ( iend >= imshape.y ) iend = imshape.y - 1;

    jinit = cx - spshape.x * ratio;
    if ( jinit < 0 ) jinit = 0;
    jend = cx + spshape.x * ratio;
    if ( jend >= imshape.x ) jend = imshape.x - 1;

    float gray = 0, x = 0, y = 0, z = 0;
    int count = 0;

    for ( int k = kinit; k < kend; k++ )
    {
        for ( int i = iinit; i < iend; i++ )
        {
            for ( int j = jinit; j < jend; j++ )
            {
                int offset = k * imshape.y * imshape.x + i * imshape.x + j;
                if ( labels[offset] == lidx ) {
                    x += j;
                    y += i;
                    z += k;
                    gray += data[offset];
                    count += 1;
                }
            }
        }
    }

    if ( count == 0 ) {
        centers[lidx].z = -1;
        centers[lidx].y = -1;
        centers[lidx].x = -1;
        return;
    } else {
        centers[lidx].f = gray / count;
        centers[lidx].x = x / count;
        centers[lidx].y = y / count;
        centers[lidx].z = z / count;
    }
}


const int dx6[6] = {-1,  0,  0,  1,  0,  0};
const int dy6[6] = { 0, -1,  0,  0,  1,  0};
const int dz6[6] = { 0,  0, -1,  0,  0,  1};

void FindNext(const int* labels, int* nlabels,
              const int3 shape, const const int lab,
              const int w, const int h, const int d,
              int* xvec, int* yvec, int* zvec,
              int* count, const int max_size)
{
    int z, y, x, ind;
    int oldlab = labels[d*shape.y*shape.x + h*shape.x + w];

    for ( int i = 0; i < 6; i++ )
    {
        z = d+dz6[i];
        y = h+dy6[i];
        x = w+dx6[i];

        if ( (z < shape.z && z >= 0) && (y < shape.y && y >= 0) &&
             (x < shape.x && x >= 0) )
        {
            ind = z*shape.y*shape.x + y*shape.x + x;
            if ( nlabels[ind] < 0 && labels[ind] == oldlab )
            {
                xvec[*count] = x;
                yvec[*count] = y;
                zvec[*count] = z;
                *count += 1;
                nlabels[ind] = lab;

                if ( *count < max_size - 1 ) {
                    FindNext(labels, nlabels, shape, lab, x, y, z,
                             xvec, yvec, zvec, count, max_size);
                }
            }
        }
    }
}

void enforceConnectivity(int* labels, const int3 shape,
                         const int max_size, const int min_size)
{
    int size = shape.z * shape.y * shape.x;
    int* nlabels = (int*)malloc(size * sizeof(int));
    memset(nlabels, -1, size * sizeof(int));

    //------------------
    // labeling
    //------------------
    int lab = 0;
    int i = 0;
    int adjlabel = 0; //adjacent label
    int* xvec = (int*)malloc(max_size * sizeof(int)); //worst case size
    int* yvec = (int*)malloc(max_size * sizeof(int)); //worst case size
    int* zvec = (int*)malloc(max_size * sizeof(int)); //worst case size

    int count = 0;

    for ( int d = 0; d < shape.z; d++ )
    {
        for ( int h = 0; h < shape.y; h++ )
        {
            for( int w = 0; w < shape.x; w++ )
            {
                int idx = d*shape.y*shape.x + h*shape.x + w;

                if ( nlabels[idx] < 0 )
                {
                    nlabels[idx] = lab;
                    //-------------------------------------------------------
                    // Quickly find an adjacent label for use later if needed
                    //-------------------------------------------------------
                    for ( int n = 0; n < 6; n++ )
                    {
                        int x = w + dx6[n];
                        int y = h + dy6[n];
                        int z = d + dz6[n];

                        if( (x >= 0 && x < shape.x) && (y >= 0 && y < shape.y) &&
                            (z >= 0 && z < shape.z) )
                        {
                            int nindex = z*shape.y*shape.x + y*shape.x + x;
                            if ( nlabels[nindex] >= 0 ) {
                                adjlabel = nlabels[nindex];
                                break;
                            }
                        }
                    }
                    xvec[0] = w; yvec[0] = h; zvec[0] = d;
                    count = 1;
                    FindNext(labels, nlabels, shape, lab, w, h, d, xvec, yvec, zvec, &count, max_size);
                    //-------------------------------------------------------
                    // If segment size is less then a limit, assign an
                    // adjacent label found before, and decrement label count.
                    //-------------------------------------------------------
                    if ( count < min_size )
                    {
                        for( int c = 0; c < count; c++ )
                        {
                            int ind = zvec[c]*shape.y*shape.x + yvec[c]*shape.x + xvec[c];
                            nlabels[ind] = adjlabel;
                        }
                        lab--;
                    }
                    lab++;
                }
                i++;
            }
        }
    }
    //------------------
    //numlabels = lab;
    //------------------
    if ( xvec ) free(xvec);
    if ( yvec ) free(yvec);
    if ( zvec ) free(zvec);

    memcpy(labels, nlabels, size * sizeof(int));

    if ( nlabels ) free(nlabels);
}


// Main function
void slicSupervoxels(const float *h_src, int *h_dest, const float compactness, \
                     const int3 im_shape, const int3 sp_shape, \
                     const int3 window, const float3 spacing, \
                     const float min_size_ratio, const float max_size_ratio, \
                     const unsigned short max_iter, const bool enforce_connectivity,
                     int gpu)
{
    // Init params
    size_t npixels = im_shape.x * im_shape.y * im_shape.z;
    size_t mem_size = sizeof(float) * npixels;
    int3 nsp = {(im_shape.x + sp_shape.x - 1) / sp_shape.x, \
                 (im_shape.y + sp_shape.y - 1) / sp_shape.y, \
                 (im_shape.z + sp_shape.z - 1) / sp_shape.z};
    size_t total = nsp.x * nsp.y * nsp.z;
    int max_xy = max(im_shape.x, im_shape.y);
    float m = compactness / (float)(max(max_xy, im_shape.z));
    m /= sqrt(im_shape.x * im_shape.y * im_shape.z / total);

    // Init cuda memory
    initCuda(gpu);

    float *d_src;
    int *d_dest;
    SLICClusterCenter *d_centers;

    hipMalloc((float **) &d_src, mem_size);
    hipMemcpy(d_src, h_src, mem_size, hipMemcpyHostToDevice);
    hipMalloc((int **) &d_dest, npixels * sizeof(int));
    hipMemset(d_dest, 0, npixels * sizeof(int));
    hipMalloc((float **) &d_centers, sizeof(SLICClusterCenter) * total);
    hipMemset(d_centers, 0, sizeof(SLICClusterCenter) * total);
    cudaCheckErrors("SRC, DST & Centers malloc");

    // bdim and gdim
    dim3 threads(1024, 1, 1);
    dim3 grid((total + 1024 - 1) / 1024, 1, 1);

    dim3 threads2(1024, 1, 1);
    dim3 grid2((npixels + 1024 - 1) / 1024, 1, 1);

    initSupervoxels<<<grid, threads>>>(d_src, d_centers, total, nsp, sp_shape, \
                                       window, im_shape);

    for ( int i = 0; i < max_iter; i++ )
    {
        assignSupervoxels<<<grid2, threads2>>>(d_src, d_centers, d_dest, m, total, spacing, nsp, sp_shape, im_shape);
        updateSupervoxels<<<grid, threads>>>(d_src, d_dest, d_centers, total, nsp, sp_shape, im_shape);
    }
    assignSupervoxels<<<grid2, threads2>>>(d_src, d_centers, d_dest, m, total, spacing, nsp, sp_shape, im_shape);

    hipMemcpy(h_dest, d_dest, npixels * sizeof(int), hipMemcpyDeviceToHost);
    cudaCheckErrors("Memcpy back");

    hipFree(d_src);
    hipFree(d_dest);
    hipFree(d_centers);
    hipDeviceReset();

    if ( enforce_connectivity ) {
        int spsize = sp_shape.x * sp_shape.y * sp_shape.z;
        int max_size = (int)(spsize * max_size_ratio);
        int min_size = (int)(spsize * min_size_ratio);
        enforceConnectivity(h_dest, im_shape, max_size, min_size);
    }
}
