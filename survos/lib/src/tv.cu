#include "hip/hip_runtime.h"
// http://gpu4vision.icg.tugraz.at/papers/2010/knoll.pdf#pub47

#include "preprocess.cuh"


__device__ __inline__
float divergence(const float* pz, const float* py, const float* px,
                 size_t idx, size_t z, size_t y, size_t x,
                 int3 shape, float3 spacing)
{
    size_t size2d = shape.y * shape.x;
    float _div = 0.0f;

    if ( z - 1 >= 0 ) {
        _div += (pz[idx] - pz[(z-1)*size2d + y*shape.x + x]) / spacing.z;
    } else {
        _div += pz[idx];
    }

    if ( y - 1 >= 0 ) {
        _div += (py[idx] - py[z*size2d + (y-1)*shape.x + x]) / spacing.y;
    } else {
        _div += py[idx];
    }

    if ( x - 1 >= 0 ) {
        _div += (px[idx] - px[z*size2d + y*shape.x + (x-1)]) / spacing.x;
    } else {
        _div += px[idx];
    }

    return _div;
}

__device__ __inline__
void gradient(const float* u, float* grad,
              size_t idx, size_t z, size_t y, size_t x,
              int3 shape, float3 spacing)
{
    size_t size2d = shape.y * shape.x;

    float uidx = u[idx];

    if ( z + 1 < shape.z ) {
        grad[0] = (u[(z+1)*size2d + y*shape.x + x] - uidx) / spacing.z;
    }

    if ( y + 1 < shape.y ) {
        grad[1] = (u[z*size2d + (y+1)*shape.x + x] - uidx) / spacing.y;
    }

    if ( x + 1 < shape.x ) {
        grad[2] = (u[z*size2d + y*shape.x + (x+1)] - uidx) / spacing.x;
    }
}


__global__
void update_u(const float* f, const float* pz, const float* py, const float* px, float* u,
              float tau, float lambda, int3 shape, float3 spacing)
{
    size_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    size_t plane = shape.y * shape.x;

    if ( idx >= plane * shape.z )
        return;

    size_t t = idx % plane;
    size_t z = idx / plane;
    size_t y = t / shape.x;
    size_t x = t % shape.x;

    float _div = divergence(pz, py, px, idx, z, y, x, shape, spacing);

    u[idx] = u[idx] * (1.0f - tau) + tau * (f[idx] + (1.0f/lambda) * _div);
}


__global__
void update_p(const float* u, float* pz, float* py, float* px,
              float tau, int3 shape, float3 spacing)
{
    size_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    size_t plane = shape.y * shape.x;

    if ( idx >= plane * shape.z )
        return;

    size_t t = idx % plane;
    size_t z = idx / plane;
    size_t y = t / shape.x;
    size_t x = t % shape.x;

    float grad[3] = {0,0,0}, q[3];
    gradient(u, grad, idx, z, y, x, shape, spacing);

    q[0] = pz[idx] + tau * grad[0];
    q[1] = py[idx] + tau * grad[1];
    q[2] = px[idx] + tau * grad[2];

    float norm = fmaxf(1.0f, sqrtf(q[0] * q[0] + q[1] * q[1] + q[2] * q[2]));

    pz[idx] = q[0] / norm;
    py[idx] = q[1] / norm;
    px[idx] = q[2] / norm;
}


// Main function
void tvdenoising(const float* src, float* dst, float lambda,
                 float3 spacing, int3 shape, int maxIter, float eps,
                 int gpu)
{
    // Init params
    size_t total = shape.x * shape.y * shape.z;
    size_t mem_size = sizeof(float) * total;

    // Init cuda memory
    int max_threads = initCuda(gpu);

    float *d_src, *d_u, *d_px, *d_py, *d_pz;

    // F
    hipMalloc(&d_src, mem_size);
    hipMemcpy(d_src, src, mem_size, hipMemcpyHostToDevice);
    cudaCheckErrors("Memory Malloc and Memset: SRC");
    // U
    hipMalloc(&d_u, mem_size);
    hipMemcpy(d_u, src, mem_size, hipMemcpyHostToDevice);
    cudaCheckErrors("Memory Malloc and Memset: U");
    // PX
    hipMalloc(&d_px, mem_size);
    hipMemset(d_px, 0, mem_size);
    cudaCheckErrors("Memory Malloc and Memset: PX");
    // PY
    hipMalloc(&d_py, mem_size);
    hipMemset(d_py, 0, mem_size);
    cudaCheckErrors("Memory Malloc and Memset: PY");
    // PZ
    hipMalloc(&d_pz, mem_size);
    hipMemset(d_pz, 0, mem_size);
    cudaCheckErrors("Memory Malloc and Memset: PZ");

    // bdim and gdim
    dim3 block(max_threads, 1, 1);
    dim3 grid((total+max_threads-1)/max_threads, 1, 1);

    float tau2, tau1;
    for ( int i = 0; i < maxIter; i++ )
    {
        tau2 = 0.3f + 0.02f * i;
        tau1 = (1.f/tau2) * ((1.f/6.f) - (5.f/(15.f+i)));

        update_u<<<grid, block>>>(d_src, d_pz, d_py, d_px, d_u, tau1, lambda,
                                  shape, spacing);

        update_p<<<grid, block>>>(d_u, d_pz, d_py, d_px, tau2,
                                  shape, spacing);
    }

    cudaCheckErrors("TV minimization");

    hipMemcpy(dst, d_u, mem_size, hipMemcpyDeviceToHost);
    cudaCheckErrors("Copy result back");

    hipFree(d_src);
    hipFree(d_u);
    hipFree(d_pz);
    hipFree(d_py);
    hipFree(d_px);
    hipDeviceReset();
}
