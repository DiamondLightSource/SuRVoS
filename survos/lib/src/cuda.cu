
#include "cuda.cuh"

#include <hipcub/hipcub.hpp>

using namespace hipcub;


bool CUDA_STARTED = false;


__host__ void initCuda()
{
    if ( CUDA_STARTED == true ) { return; }

    int devID = 0;
    hipError_t error;
    hipDeviceProp_t deviceProp;

    error = hipGetDevice(&devID);

    if (error != hipSuccess)
    {
        printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
    }

    error = hipGetDeviceProperties(&deviceProp, devID);

    if (deviceProp.computeMode == hipComputeModeProhibited)
    {
        fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
        exit(EXIT_SUCCESS);
    }

    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
    }
    else
    {
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    }

    CUDA_STARTED = true;
}


template<typename T> __host__
T reduce(const T* h_in, size_t num_items)
{
    initCuda();

    CachingDeviceAllocator  g_allocator(true);

    T sum;

    // Allocate problem device arrays
    T *d_in = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_in, sizeof(T) * num_items));
    // Initialize device input
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(T) * num_items, hipMemcpyHostToDevice));
    // Allocate device output array
    T *d_out = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_out, sizeof(T) * 1));

    // Request and allocate temporary storage
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    CubDebugExit(DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items));
    CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));
    // Run
    CubDebugExit(DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items));

    // Copy the result to host
    CubDebugExit(hipMemcpy(&sum, d_out, sizeof(T) * 1, hipMemcpyDeviceToHost));

    if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
    if (d_out) CubDebugExit(g_allocator.DeviceFree(d_out));
    if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));

    return sum;
}

template long reduce(const long *h_in, size_t num_items);
template int reduce(const int *h_in, size_t num_items);
template float reduce(const float *h_in, size_t num_items);
template double reduce(const double *h_in, size_t num_items);


template<typename T> __host__
T reduceMax(const T* h_in, size_t num_items)
{
    initCuda();

    CachingDeviceAllocator  g_allocator(true);

    T tmax;

    // Allocate problem device arrays
    T *d_in = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_in, sizeof(T) * num_items));
    // Initialize device input
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(T) * num_items, hipMemcpyHostToDevice));
    // Allocate device output array
    T *d_out = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_out, sizeof(T) * 1));

    // Request and allocate temporary storage
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    CubDebugExit(DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items));
    CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));
    // Run
    CubDebugExit(DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items));

    // Copy the result to host
    CubDebugExit(hipMemcpy(&tmax, d_out, sizeof(T) * 1, hipMemcpyDeviceToHost));

    if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
    if (d_out) CubDebugExit(g_allocator.DeviceFree(d_out));
    if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));

    return tmax;
}

template long reduceMax(const long *h_in, size_t num_items);
template int reduceMax(const int *h_in, size_t num_items);
template float reduceMax(const float *h_in, size_t num_items);
template double reduceMax(const double *h_in, size_t num_items);